#include "hip/hip_runtime.h"
/*
 * Mesh.cpp
 *
 *  Created on: 27 lis 2019
 *      Author: karol
 */

#include "Mesh.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include "DisplayCalculatorKernels.h"
#include "mat4x4.h"

Mesh::Mesh() {
	worldMatrix = getIdentityMatrix();

}

Mesh::~Mesh() {
	if(points != nullptr)
	{
		delete[] points;
	}
	if(triangles != nullptr)
	{
		delete[] triangles;
	}

}
void Mesh::SetPoints(float3 points[], int length)
{
	if(this->points != nullptr)
	{
		delete[] this->points;
	}
	this->points = new float3[length];
	for(int i = 0; i < length; i++)
	{
		this->points[i] = points[i];
	}
	pointsLength = length;
	initialized = false;
}
void Mesh::SetTriangles(int triangles[], int length)
{
	if(this->points == nullptr)
	{
		throw "Initialize points first";
	}
	if(length % 3 != 0)
	{
		throw "length of triangles array must be divisble by 3";
	}
	if(this->triangles != nullptr)
	{
		delete[] this->triangles;
	}
	this->triangles = new short[length];
	for(int i = 0; i < length; i++)
	{
		if(pointsLength <= triangles[i])
			throw "Incorrect triangle: points array doesn't have this many numbers";
		this->triangles[i] = (short)triangles[i];
	}
	trianglesLength = length;
}
void Mesh::CopyToDevice()
{
	if(this->triangles == nullptr)
	{
		throw "Initialize triangles, points  first";
	}
	if(this->d_points != nullptr)
	{
		hipFree(d_points);
	}
	if(this->d_points_transformed != nullptr)
	{
		hipFree(d_points_transformed);
	}
	hipMalloc(&d_points, sizeof(float3)*pointsLength);
	getLastCudaError("couldn't malloc d_points in mesh");

	hipMalloc(&d_points_transformed, sizeof(float3)*pointsLength);
	getLastCudaError("couldn't malloc d_points_transformed in mesh");

	hipMemcpy(d_points, points, sizeof(float3)*pointsLength, hipMemcpyHostToDevice);
	getLastCudaError("couldn't memcpy points in mesh");

	SaveToConstantMemory(triangles, pointsLength, trianglesLength);
	getLastCudaError("Couldn't memcpy data to constant for Mesh");
	initialized = true;
}
void Mesh::UpdateMeshVertices()
{
	worldMatrix.multiplyAllVectors(d_points, d_points_transformed, pointsLength);
	SaveVerticesToConstantMemory(d_points_transformed, pointsLength);
}

bool Mesh::IsInitialized() {
	return initialized;
}

DeviceMeshData Mesh::GetDeviceMeshData() {
	DeviceMeshData res;
	res.pointsLength = pointsLength;
	res.trianglesLength = trianglesLength;

	return res;
}
void Mesh::SetWorldMatrix(mat4x4 matrix)
{
	worldMatrix = matrix;
}
