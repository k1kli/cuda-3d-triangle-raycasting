#include "hip/hip_runtime.h"
/*
 * Mesh.cpp
 *
 *  Created on: 27 lis 2019
 *      Author: karol
 */

#include "Mesh.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include "DisplayCalculatorKernels.h"
#include "mat4x4.h"

Mesh::Mesh(bool onCPU) {
	worldMatrix = getIdentityMatrix();
	this->onCPU = onCPU;

}

Mesh::~Mesh() {
	if(points != nullptr)
	{
		delete[] points;
		if(onCPU)
			delete[] this->cpu_points_transformed;
	}
	if(triangles != nullptr)
	{
		delete[] triangles;
	}
	if(this->d_points != nullptr)
	{
		hipFree(d_points);
	}
	if(this->d_points_transformed != nullptr)
	{
		hipFree(d_points_transformed);
	}
	if(this->d_triangles != nullptr)
	{
		hipFree(d_triangles);
	}

}
void Mesh::SetPoints(float3 points[], int length)
{
	if(this->points != nullptr)
	{
		delete[] this->points;
		this->points = nullptr;
		if(onCPU)
		{
			delete[] this->cpu_points_transformed;
			this->cpu_points_transformed = nullptr;
		}
	}
	this->points = new float3[length];
	if(onCPU)
		this->cpu_points_transformed = new float3[length];
	for(int i = 0; i < length; i++)
	{
		this->points[i] = points[i];
	}
	pointsLength = length;
	initialized = false;
}
void Mesh::SetTriangles(int triangles[], int length)
{
	if(this->points == nullptr)
	{
		throw "Initialize points first";
	}
	if(length % 3 != 0)
	{
		throw "length of triangles array must be divisble by 3";
	}
	if(this->triangles != nullptr)
	{
		delete[] this->triangles;
		this->triangles = nullptr;
	}
	this->triangles = new int[length];
	for(int i = 0; i < length; i++)
	{
		if(pointsLength <= triangles[i])
			throw "Incorrect triangle: points array doesn't have this many numbers";
		this->triangles[i] = triangles[i];
	}
	trianglesLength = length;
	if(onCPU) initialized = true;
}
void Mesh::CopyToDevice()
{
	if(onCPU)
		throw "You shouldn't do this on cpu";
	if(this->triangles == nullptr)
	{
		throw "Initialize triangles, points  first";
	}
	if(this->d_points != nullptr)
	{
		hipFree(d_points);
		d_points = nullptr;
	}
	if(this->d_points_transformed != nullptr)
	{
		hipFree(d_points_transformed);
		d_points_transformed = nullptr;
	}
	if(this->d_triangles != nullptr)
	{
		hipFree(d_triangles);
		d_triangles = nullptr;
	}
	hipMalloc(&d_points, sizeof(float3)*pointsLength);
	getLastCudaError("couldn't malloc d_points in mesh");

	hipMalloc(&d_points_transformed, sizeof(float4)*pointsLength);
	getLastCudaError("couldn't malloc d_points_transformed in mesh");

	hipMalloc(&d_triangles, sizeof(int)*trianglesLength);
	getLastCudaError("couldn't malloc d_triangles in mesh");

	hipMemcpy(d_points, points, sizeof(float3)*pointsLength, hipMemcpyHostToDevice);
	getLastCudaError("couldn't memcpy points in mesh");

	hipMemcpy(d_triangles, triangles, sizeof(int)*trianglesLength, hipMemcpyHostToDevice);
	getLastCudaError("couldn't memcpy triangles in mesh");

	SaveToConstantMemory();
	getLastCudaError("Couldn't save data to constant for Mesh");
	initialized = true;
}
void Mesh::UpdateMeshVertices()
{
	if(onCPU)
	{
		worldMatrix.multiplyAllCPUVectors(points, cpu_points_transformed, pointsLength);
	}
	else
	{
		worldMatrix.multiplyAllVectors(d_points, d_points_transformed, pointsLength);
	}
}

bool Mesh::IsInitialized() {
	return initialized;
}

DeviceMeshData Mesh::GetDeviceMeshData() {
	if(onCPU)
		throw "You shouldn't do this on cpu";
	DeviceMeshData res;
	res.pointsLength = pointsLength;
	res.trianglesLength = trianglesLength;
	res.d_points = d_points_transformed;
	res.d_triangles = d_triangles;

	return res;
}
void Mesh::SetWorldMatrix(mat4x4 matrix)
{
	worldMatrix = matrix;
}
